#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void bcast(int arg) 
{
    int laneId = threadIdx.x & 0x1f;
    int value;

    if (laneId == 0)  value = arg; 

    // Synchronize all threads in warp, and get "value" from lane 0
    value = __shfl_sync(0xffffffff, value, 0);
    if (value != arg)
        printf("Thread %d failed.\n", threadIdx.x);
}

int main()
{
    bcast<<< 1, 32 >>>(1234);
    hipDeviceSynchronize();

    return 0;
}

#include "hip/hip_runtime.h"

/* asum: sum of all entries of a vector.
 * This code only calculates one block to show the usage of shared memory and synchronization */

#include <stdio.h>
#include <hip/hip_runtime.h>

/* cg */
#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;

typedef double FLOAT;

__device__ void warpReduce(volatile FLOAT *sdata, int tid)
{
    sdata[tid] += sdata[tid + 32];
    sdata[tid] += sdata[tid + 16];
    sdata[tid] += sdata[tid + 8];
    sdata[tid] += sdata[tid + 4];
    sdata[tid] += sdata[tid + 2];
    sdata[tid] += sdata[tid + 1];
}

__global__ void SumReduceKnl(const FLOAT *x, FLOAT *y)
{
    __shared__ FLOAT sdata[256];
    int tid = threadIdx.x;
    thread_block g = this_thread_block();

    /* load data to shared mem */
    sdata[tid] = x[tid];
    g.sync();

    /* reduction using shared mem */
    if (tid < 128) sdata[tid] += sdata[tid + 128];
    g.sync();

    if (tid < 64) sdata[tid] += sdata[tid + 64];
    g.sync();

    if (tid < 32) warpReduce(sdata, tid);
    g.sync();

    if (tid == 0) y[0] = sdata[0];
}

int main()
{
    int N = 256;   /* must be 256 */
    int nbytes = N * sizeof(FLOAT);

    FLOAT *dx = NULL, *hx = NULL;
    FLOAT *dy = NULL;
    int i;
    FLOAT as = 0;

    /* allocate GPU mem */
    hipMalloc((void **)&dx, nbytes);
    hipMalloc((void **)&dy, sizeof(FLOAT));

    if (dx == NULL || dy == NULL) {
        printf("couldn't allocate GPU memory\n");
        return -1;
    }

    printf("allocated %e MB on GPU\n", nbytes / (1024.f * 1024.f));

    /* alllocate CPU mem */
    hx = (FLOAT *) malloc(nbytes);

    if (hx == NULL) {
        printf("couldn't allocate CPU memory\n");
        return -2;
    }
    printf("allocated %e MB on CPU\n", nbytes / (1024.f * 1024.f));

    /* init */
    for (i = 0; i < N; i++) {
        hx[i] = 1;
    }

    /* copy data to GPU */
    hipMemcpy(dx, hx, nbytes, hipMemcpyHostToDevice);

    /* call GPU */
    SumReduceKnl<<<1, N>>>(dx, dy);

    /* let GPU finish */
    hipDeviceSynchronize();

    /* copy data from GPU */
    hipMemcpy(&as, dy, sizeof(FLOAT), hipMemcpyDeviceToHost);

    printf("SumReduceKnl, answer: 256, calculated by GPU:%g\n", as);

    hipFree(dx);
    hipFree(dy);
    free(hx);

    return 0;
}
